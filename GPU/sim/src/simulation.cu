#include "hip/hip_runtime.h"
// this file will manage the simulation, handling CPU GPU communication, running the phases, and managing memory
#include "config.h"
#include <stdio.h>
#include "simulation.cuh"
#include <hiprand/hiprand_kernel.h>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <string>
#include <atomic>
#include <iostream>

//THIS CODE SPONSORED BY SHADOW WIZARD MONEY GANG
//"We love casting spells"

__host__ void simulation() {

        //Sim Variables
        //cell_grid[value] = ID of person in cell
        int* cell_grid_first = new int[GRID_SIZE * GRID_SIZE];
        int* cell_grid_last = new int[GRID_SIZE * GRID_SIZE];
        //next[value] = ID of next person in cell, 0 if none
        int* next = new int[(POPULATION)];
        //array of variants
        int variant_count = 0;
        int variant_cap = 64;
        Variant* variants = new Variant[variant_cap]; 

        //Person Variables, SoA style
        //x position of each person, may not be needed
        int* positions = new int[(POPULATION)];
        //variant[value] = ID of variant, negative value if not infected
        int* variant = new int[(POPULATION)];
        //time remaining until immunity expires, negative if no longer immune
        int* immunity = new int[(POPULATION)];
        // whether the person is dead, 0 or positive is alive and negative is dead
        // also used to keep track of infection time where value of alive keeps track of infection period
        int* dead = new int[(POPULATION)]; 
        //TODO: handle the case of preventing fresh infections from infecting others
        //current implementation is naive, but works (at a 4 byte cost per person)
        bool* fresh = new bool[(POPULATION)];

        int sim_bytes_used = 0;
        int people_bytes_used = 0;
        sim_bytes_used += sizeof(int) * GRID_SIZE * GRID_SIZE * 2; //cell_grid_first, cell_grid_last
        sim_bytes_used += sizeof(int) * GRID_SIZE * GRID_SIZE; //next
        sim_bytes_used += sizeof(Variant) * variant_cap; //variants
        people_bytes_used += sizeof(int) * (POPULATION) * 4; //positions, variant, immunity, dead
        people_bytes_used += sizeof(bool) * (POPULATION); //fresh
        printf("Sim memory footprint: %d bytes\n", sim_bytes_used);
        printf("People memory footprint: %d bytes\n", people_bytes_used);

        //initialize on host
        printf("Initializing data\n");

        //zero out SoA
        for (int i = 0; i < POPULATION; i++) {
            positions[i] = 0;
            variant[i] = -1;
            immunity[i] = -1;
            dead[i] = 0;
            fresh[i] = false;
        }

        //place 2 people in the sim together
        positions[0] = 128 * 256 + 128;
        positions[1] = 128 * 256 + 128;

        //set up a variant
        variant_count = 1;
        Variant v {};
        v.id = 0,
        v.recovery_time = 2;
        v.mortality_rate = 0.0;
        v.infection_rate = 1.0;
        v.mutation_rate = 1.0;
        v.immunity_time = 2;
        
        variants[0] = v;

        //infect the first person with the variant
        variant[0] = 0;
        dead[0] = v.recovery_time;

        //infect someone else with the variant
        variant[2] = 0;
        dead[2] = v.recovery_time;

        //show initial values
        printf("Initial Values:\n");
        for (int i = 0; i < POPULATION; i++) {
            printf("Person %d is at position (%d, %d). Fresh: %d Variant: %d, Immunity: %d, Dead: %d\n", i, positions[i] % GRID_SIZE, positions[i] / GRID_SIZE, fresh[i], variant[i], immunity[i], dead[i]);
        }
        printf("==============================================================================\n");

        //set up GPU memory
        int *d_cell_grid_first = NULL;
        int *d_cell_grid_last = NULL;
        int *d_next = NULL;
        int *d_variant_count = NULL;
        int *d_variant_cap = NULL;
        Variant *d_variants = NULL;

        int *d_position = NULL;
        int *d_variant = NULL;
        int *d_immunity = NULL;
        int* d_dead = NULL;
        bool* d_fresh = NULL;

        printf("Allocating GPU memory\n");
        hipMalloc((void**)&d_cell_grid_first, sizeof(int) * GRID_SIZE * GRID_SIZE);
        hipMalloc((void**)&d_cell_grid_last, sizeof(int) * GRID_SIZE * GRID_SIZE);
        hipMalloc((void**)&d_next, sizeof(int) * GRID_SIZE * GRID_SIZE);
        hipMalloc((void**)&d_variant_count, sizeof(int));
        hipMalloc((void**)&d_variant_cap, sizeof(int));

        hipMalloc((void**)&d_variants, sizeof(Variant) * variant_cap);
        hipMalloc((void**)&d_position, sizeof(int) * (POPULATION));
        hipMalloc((void**)&d_variant, sizeof(int) * (POPULATION));
        hipMalloc((void**)&d_immunity, sizeof(int) * (POPULATION));
        hipMalloc((void**)&d_dead, sizeof(int) * (POPULATION));
        hipMalloc((void**)&d_fresh, sizeof(bool) * (POPULATION));
        cudaCheck("Error allocating GPU memory");

        printf("Copying data to GPU\n");
        hipMemcpy(d_cell_grid_first,   cell_grid_first, sizeof(int) * GRID_SIZE * GRID_SIZE,   hipMemcpyHostToDevice);
        hipMemcpy(d_cell_grid_last,    cell_grid_last, sizeof(int) * GRID_SIZE * GRID_SIZE,    hipMemcpyHostToDevice);
        hipMemcpy(d_next, next,        sizeof(int) * GRID_SIZE * GRID_SIZE,                    hipMemcpyHostToDevice);
        hipMemcpy(d_variant_count,     &variant_count, sizeof(int),                            hipMemcpyHostToDevice);
        hipMemcpy(d_variant_cap,       &variant_cap, sizeof(int),                              hipMemcpyHostToDevice);
        hipMemcpy(d_variants,          variants, sizeof(Variant) * variant_cap,                hipMemcpyHostToDevice);

        hipMemcpy(d_position,          positions, sizeof(int) * (POPULATION),                  hipMemcpyHostToDevice);
        hipMemcpy(d_variant,           variant, sizeof(int) * (POPULATION),                    hipMemcpyHostToDevice);
        hipMemcpy(d_immunity,          immunity, sizeof(int) * (POPULATION),                   hipMemcpyHostToDevice);
        hipMemcpy(d_dead, dead,        sizeof(int) * (POPULATION),                             hipMemcpyHostToDevice);
        hipMemcpy(d_fresh, fresh,      sizeof(bool) * (POPULATION),                            hipMemcpyHostToDevice);
        cudaCheck("Error copying data to GPU");

        // run the gpu code once
        for(int i = 0; i < EPOCHS; i++){
            printf("Epoch %d =====================================================================\n", i + 1);
            printf("movePeople\n");
            //movePeople<<<MOVE_BLOCKS, MOVE_THREADS>>>(d_position, i);
            hipDeviceSynchronize();
            cudaCheck("movePeople error");
            printf("infectPeople\n");
            infectPeople<<<INFECTION_BLOCKS, INFECTION_THREADS>>>(d_variants, d_position, d_variant_count, d_variant_cap, d_variant, d_immunity, d_dead, d_fresh);
            hipDeviceSynchronize();
            cudaCheck("infectPeople error");
            //TODO: zero the dead array
            //printf("zeroDead");
            //zeroDead<<<ZERO_DEAD_BLOCKS, ZERO_DEAD_THREADS>>>(d_dead);
            printf("killPeople\n");
            killPeople<<<KILL_BLOCKS,KILL_THREADS>>>(d_variants, d_variant, d_dead, d_fresh);
            hipDeviceSynchronize();
            cudaCheck("killPeople error");
            printf("tick\n");
            tick<<<TICK_BLOCKS,TICK_THREADS>>>(d_variants, d_immunity, d_variant, d_dead, d_fresh);
            hipDeviceSynchronize();
            cudaCheck("tick error");
            gpuPeek<<<1, 1>>>(d_position, d_variant, d_immunity, d_dead, d_fresh);
            hipDeviceSynchronize();
            cudaCheck("gpuPeek error");
            showVariants<<<1, 1>>>(d_variants, d_variant_count);
            hipDeviceSynchronize();
            cudaCheck("showVariants error");
            //zero the fresh array
            hipMemset(d_fresh, 0, POPULATION*sizeof(bool));
            cudaCheck("gpuPeek error");
            

        }
        printf("Epochs complete\n");

        printf("Copying data back to CPU\n");
        hipMemcpy(cell_grid_first,     d_cell_grid_first, sizeof(int) * GRID_SIZE * GRID_SIZE, hipMemcpyDeviceToHost);
        hipMemcpy(cell_grid_last,      d_cell_grid_last, sizeof(int) * GRID_SIZE * GRID_SIZE,  hipMemcpyDeviceToHost);
        hipMemcpy(next, d_next,        sizeof(int) * GRID_SIZE * GRID_SIZE,                    hipMemcpyDeviceToHost); 
        hipMemcpy(&variant_cap,        d_variant_cap, sizeof(int),                             hipMemcpyDeviceToHost);
        hipMemcpy(variants,            d_variants, sizeof(Variant) * variant_cap,              hipMemcpyDeviceToHost);

        hipMemcpy(positions,           d_position, sizeof(int) * (POPULATION),                 hipMemcpyDeviceToHost);
        hipMemcpy(variant,             d_variant, sizeof(int) * (POPULATION),                  hipMemcpyDeviceToHost);
        hipMemcpy(immunity,            d_immunity, sizeof(int) * (POPULATION),                 hipMemcpyDeviceToHost);
        hipMemcpy(dead,                d_dead, sizeof(int) * (POPULATION),                     hipMemcpyDeviceToHost);
        hipMemcpy(fresh,               d_fresh, sizeof(bool) * (POPULATION),                   hipMemcpyDeviceToHost);
        cudaCheck("Error copying data back to CPU");

        //print out the positions of all the people
        printf("====================================\n");
        printf("Final Values:\n");
        for (int i = 0; i < POPULATION; i++) {
            printf("Person %d is at position (%d, %d). Fresh: %d Variant: %d, Immunity: %d, Dead: %d\n", i, positions[i] % GRID_SIZE, positions[i] / GRID_SIZE, fresh[i], variant[i], immunity[i], dead[i]);
        }

        printf("Freeing GPU memory\n");
        hipFree(d_cell_grid_first);
        hipFree(d_cell_grid_last);
        hipFree(d_next);
        hipFree(d_variant_count);
        hipFree(d_variant_cap);
        hipFree(d_variants);

        hipFree(d_position);
        hipFree(d_variant);
        hipFree(d_immunity);
        hipFree(d_dead);
        hipFree(d_fresh);
        cudaCheck("Error freeing GPU memory");
}

__global__ void showVariants(Variant* variants, int * variant_count){
    //expected to be 1 thread, 1 block
    printf("Current Variants =====================================================================\n");
    for (int i = 0; i < *variant_count; i++) {
        //use the Variant.toString() method
        printf("Variant %d - mort: %f, inf: %f, mut: %f, rec: %d, imm: %d\n", i, variants[i].mortality_rate, variants[i].mutation_rate, variants[i].immunity_time, variants[i].recovery_time, variants[i].immunity_time);
    }
}

__global__ void gpuPeek(int* positions, int* variant, int* immunity, int* dead, bool* fresh){
    //expected to be 1 thread, 1 block
    for (int i = 0; i < POPULATION; i++) {
        printf("Person %d is at position (%d, %d). Fresh: %d Variant: %d, Immunity: %d, Dead: %d\n", i, positions[i] % GRID_SIZE, positions[i] / GRID_SIZE, fresh[i], variant[i], immunity[i], dead[i]);
    }
}

//shortens the cuda error checking code to one line whereever it is called
void cudaCheck(const std::string &message){
    if (hipGetLastError() != hipSuccess){
        std::cout << message << std::endl;
        return;
    }
}
//TODO: replace two kernel calls with one, and just barrier sync
// move people randomly around the grid
__global__ void movePeople(int *positions, int epoch) {
    //move every person in the grid a random amount in each direction
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    if(tid > POPULATION){
        return;
    }
    //iterate over people, move them
    for(int i = tid; i < POPULATION; i += stride){
        int position = positions[i];

        //TODO: do this with only one random number
        //get movements, adding in the epoch to make sure they are different
        //add in the total number of threads to make sure they are different for rand_y
        int rand_x = randomMovement(tid + epoch);
        int rand_y = randomMovement(tid + epoch + (threadIdx.x * blockDim.x * gridDim.x));
        printf("Person %d is moving (%d, %d)", i, rand_x, rand_y);
        //add the movements back into position
        int x = position % GRID_SIZE;
        int y = position / GRID_SIZE;

        //move, wrap, and stay positive
        x = (x + rand_x + GRID_SIZE) % GRID_SIZE;
        y = (y + rand_y + GRID_SIZE) % GRID_SIZE;

        position = y * GRID_SIZE + x;
        positions[i] = position;
    }
}

//this function will update the cell_grids and next arrays
__global__ void updateCellOccupancy(int *cell_grid_first, int *cell_grid_last, int *positions, int *next){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    if (tid >= POPULATION){
        return;
    }
    //iterate over people, updating the cell grid for each person
    for(int i = tid; i < POPULATION; i += stride){
        //get the cell they should be in
        int cell = positions[i];

        //check if that cell is occupied, if not, occupy it (use atomics)
        if(atomicCAS(&cell_grid_first[cell], -1, i) == -1){
            //was not occupied, cell_grid_first[cell] is now i
            cell_grid_last[cell] = i;
        } else {
            while(1){
                //try to CAS the next[last] to our index
                if(atomicCAS(&next[cell_grid_last[cell]], -1, i) == -1){
                    //it worked, which means next[last] is now i
                    //now update cell_grid_last to be i
                    //and set next[i] to -1
                    //other threads will fail to CAS, so no other thread will touch these values
                    cell_grid_last[cell] = i;
                    next[i] = -1;
                    break;
                }
            }
        }
    }
}

//this function will update the infection status based on people sharing cells
__global__ void infectPeople(Variant* variants, int* positions, int *variant_count, int *variant_cap, int* variant, int* immunity, int* dead, bool* fresh) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    //handle threads > population
    if (tid >= POPULATION){
        printf("Unused thread\n");
        return;
    }

    for (int src = tid; src < POPULATION; src += stride) {
        //get source person details in outer loop
        Variant src_variant = variants[variant[tid]];
        int src_pos = positions[src];
        //dead do not infect
        if(dead[src] < 0){
            printf("T%d - Ignore %d, dead\n", tid, src);
            continue;
        }
        //uninfected do not infect
        if(variant[src] < 0){
            printf("T%d - Ignore %d, uninfected\n", tid, src);
            continue;
        }
        //fresh infections do not infect
        if(fresh[src]){
            printf("T%d - Ignore %d, fresh\n", tid, src);
            continue;
        }
        for (int dst_offset = 0; dst_offset < POPULATION; dst_offset++) {
            int dst = (src + tid + dst_offset) % POPULATION;
            int dst_pos = positions[dst];
            //ignore immune
            if(immunity[dst] > 0){
                printf("T%d - %d Ignore %d, immune\n", tid, src, dst);
                continue;
            }
            //ignore already infected
            if(variant[dst] >= 0){
                printf("T%d - %d Ignore %d, already infected\n", tid, src, dst);
                continue;
            }
            //ignore dead
            if(dead[dst] < 0){
                printf("T%d - %d Ignore %d, dead\n", tid, src, dst);
                continue;
            }
            //ignore self
            if(src == dst){
                printf("T%d - %d Ignore %d, self\n", tid, src, dst);
                continue;
            }
            printf("T%d - Checking: %d and %d\n", tid, src, dst);
            //check if cell shared
            if(src_pos == dst_pos){
                printf("T%d - Pos Match: %d to %d\n", tid, src, dst);
                //check for infection
                if(randomFloat(dst) < src_variant.infection_rate){
                    //check for mutation
                    if(randomFloat(dst) < src_variant.mutation_rate){
                        //atomicCAS the fresh infection to true
                        if(atomicCAS(&fresh[dst], 0, 1) == 0){
                            //give new variant to dst person
                            printf("T%d - Mutation: P%d to P%d\n", tid, src, dst);
                            int dst_variant = createVariant(variants, variant_count, variant_cap, variant[src]);
                            variant[dst] = dst_variant;
                            dead[dst] = variants[dst_variant].recovery_time;
                        } else {
                            printf("T%d - %d Ignore %d, contended\n", tid, src, dst);
                        } //else, someone else got there first
                    } else {
                        //atomicCAS the fresh infection to true
                        if(atomicCAS(&fresh[dst], 0, 1) == 0){
                            //give same variant to dst person
                            printf("T%d - Infection: P%d to P%d\n", tid, src, dst);
                            variant[dst] = variant[src];
                            dead[dst] = variants[variant[src]].recovery_time;
                        } else {
                            printf("T%d - %d Ignore %d, contended\n", tid, src, dst);
                        } //else, someone else got there first
                    }
                } //lucky them
            }
        }
    }
}

//takes a pointer to a float and overwrites it with a new float +- MUTATION_RANGE% of the original
//no if statements
__device__ float mutate_helper(float &original, int seed) {
    float rand_percent = 2 * randomFloat(seed) - 1;
    //edit the original value by a random percentage
    original *= 1 + rand_percent * MUTATION_RANGE;
    original = max(original, 0.0f);
    original = min(original, 1.0f);
}

__device__ int int_mutate_helper(int &original, int seed){
    //due to int rounding, small values will get "stuck" at 1 or 0 so minimum mutation is is +- 1 for ints
    float rand_percent = 2 * randomFloat(seed) - 1;
    float mutation = rand_percent * MUTATION_RANGE * original;
    //edit the original value by a random percentage
    if(abs(mutation) < 1){
        mutation = mutation > 0 ? 1 : -1;
    }
    original += mutation;
    original = max(original, 1);
}

//device function to create a variant
__device__ int createVariant(Variant *variants, int *variant_count, int *variant_cap, int source_variant) {
    printf("Variant created: %d\n", *variant_count);
    printf("====================================\n");
    printf("Old variant data");
    printf("id: %d\n", variants[source_variant].id);
    printf("mutation_rate: %f\n", variants[source_variant].mutation_rate);
    printf("infection_rate: %f\n", variants[source_variant].infection_rate);
    printf("mortality_rate: %f\n", variants[source_variant].mortality_rate);
    printf("recovery_time: %d\n", variants[source_variant].recovery_time);
    printf("immunity_time: %d\n", variants[source_variant].immunity_time);
    printf("====================================\n");
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //check if we need to reallocate the variants array    
    if(*variant_count == *variant_cap){
        printf("Resizing variants\n");
        *variant_cap *= 2;
        //realloc not allowed, but memcpy and malloc are
        Variant *new_variants = (Variant*)malloc(sizeof(Variant) * (*variant_cap));
        //copy the old data over
        memcpy(new_variants, variants, sizeof(Variant) * (*variant_cap));
        //free the old data
        free(variants);
        //set the variants pointer to the new data
        variants = new_variants;
    }

    //create the new variant, copy the old variant
    Variant new_variant = variants[source_variant];
    new_variant.id = *variant_count;
    //change the parameters of the new variant by up to MUTATION_RANGE
    mutate_helper(new_variant.mortality_rate, tid + 1);
    mutate_helper(new_variant.infection_rate, tid + 2);
    mutate_helper(new_variant.mutation_rate, tid + 3);
    //cap the floats at 0 and 1
    int_mutate_helper(new_variant.recovery_time, tid + 4);
    int_mutate_helper(new_variant.immunity_time, tid + 5);
    //put this variant in the variants array, increment the variant count, and return the index of the new variant
    int new_variant_index = atomicAdd(variant_count, 1);
    variants[new_variant_index] = new_variant;
    printf("====================================\n");
    printf("New variant data");
    printf("id: %d\n", new_variant.id);
    printf("mutation_rate: %f\n", new_variant.mutation_rate);
    printf("infection_rate: %f\n", new_variant.infection_rate);
    printf("mortality_rate: %f\n", new_variant.mortality_rate);
    printf("recover_time: %d\n", new_variant.recovery_time);
    printf("immunity_time: %d\n", new_variant.immunity_time);
    printf("====================================\n");
    return new_variant_index;
}

// Kills people based on variant mortality rate
__global__ void killPeople(Variant* variants, int* variant, int* dead, bool* fresh) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= POPULATION){
        return;
    }

    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < POPULATION; i += stride) {
        if (variant[i] < 0 || fresh[i]){
            // Uninfected, cannot kill
            return;
        }

        // Get the variant of the person
        Variant our_variant = variants[variant[i]];
        

        // Roll die to determine if killed off
        if (randomFloat(tid) < our_variant.mortality_rate) { 
            dead[i] = -1; // Mark as dead
        }
    }
}

// Ticks immunity and infection times for individuals
__global__ void tick(Variant* variants, int* immunity, int* variant, int* dead, bool* fresh) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= POPULATION){
        return;
    }

    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < POPULATION; i += stride) {

        if (dead[i] < 0) {
            // dead, cannot tick
            return;
        }

        // Tick immunity time
        immunity[i] = ::max(--immunity[i], -1);

        //currently infected but survived, don't tick fresh
        if (dead[i] > 0) {
            // Tick infection time
            if(fresh[i] == false){
                dead[i] = ::max(--dead[i], 0);
            }
            return;
        }

        //either recovering, or were never infected
        //check if variant is > 0 to see if infected
        if (variant[i] >= 0) {
            printf("T%d - Recovered: P%d\n", tid, i);
            // Gain immunity
            immunity[i] = variants[variant[i]].immunity_time;
            // Mark as uninfected
            variant[i] = -1;
        }
    }
}

// device function to make a random movement
__device__ int randomMovement(int thread_id) {
    hiprandState_t state;
    hiprand_init(RANDOM_SEED, thread_id, 0, &state);
    //make it between -RANGE and RANGE
    return hiprand_uniform(&state) * (MOVE_RANGE * 2) - MOVE_RANGE;
}

// device function to make a random float between 0 and 1
__device__ float randomFloat(int thread_id) {
    hiprandState_t state;
    hiprand_init(RANDOM_SEED, thread_id, 0, &state);
    float result = hiprand_uniform(&state);
    //printf("Random float: %f\n", result);
    return result;
}

//TODO: test inline
__host__ __device__ int coordToIndex(int x, int y) {
    return x * GRID_SIZE + y;
}